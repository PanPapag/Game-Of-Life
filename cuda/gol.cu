#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../common/headers/io_utils.h"
#include "../common/headers/utils.h"

#define BLOCK_SIZE 4

extern program_options options;

__device__ char get_cell_inbounds(int x, int y, int size, char* grid) {
  if (x >= 0 && x < size && y >= 0 && y < size) {
    return grid[x * size + y];
  }
  return '0';
}

__device__ int count_alive_neighbours(int x, int y, int size, char* grid) {
  int alive_neighbours = (get_cell_inbounds(x-1, y, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y, size, grid) == '1') +
                         (get_cell_inbounds(x, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x, y+1, size, grid) == '1') +
                         (get_cell_inbounds(x-1, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x-1, y+1, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y+1, size, grid) == '1');
  return alive_neighbours;
}

__device__ void apply_game_rules(int index, char* cur_grid, char* next_grid, int alive_neighbours) {
  if (cur_grid[index] == '1') {
    // 0 or 1 neighbours -> the cell dies
    if (alive_neighbours < 2) {
      next_grid[index] = '0';
    }
    // 2 or 3 neighbours -> the cell survives
    else if (alive_neighbours < 4) {
      next_grid[index] = '1';
    }
    // more than 4 neighbours -> the cell dies due to overpopulation
    else {
      next_grid[index] = '0';
    }
  }
  // rules regarding dead cells
  else {
    // exactly 3 neighbours -> a new cell is born
    if (alive_neighbours == 3) {
      next_grid[index] = '1';
    }
  }
}

__global__ void evolution(char* cur_grid, char* next_grid, int size) {
  int x = blockDim.x * BLOCK_SIZE + threadIdx.x;
  int y = blockDim.y * BLOCK_SIZE + threadIdx.y;
  int index = x * size + y;

  int alive_neighbours = count_alive_neighbours(x, y, size, cur_grid);

  apply_game_rules(index, cur_grid, next_grid, alive_neighbours);
}

void read_input(const char* input_file, char* grid) {
  int i = 0;
  char c;
  while ((c = getchar()) != '\n') {
    grid[i++] = c;
  }
}

int main(int argc, char* argv[]) {
  char* h_grid;            // Grid on host
  char* d_grid;            // Grid on device
  char* d_next_gen_grid;   // Next generation grid used on device
  char* d_tmp_grid;        // tmp grid pointer used to switch between grid and next_gen_grid

  parse_command_line_arguments(argc, argv);

  size_t grid_bytes = options.size * options.size * sizeof(char);

  // Allocate memory for host grid
  h_grid = (char*) malloc(grid_bytes);

  // Allocate memory for device grids
  hipMalloc(&d_grid, grid_bytes);
  hipMalloc(&d_next_gen_grid, grid_bytes);

  // Read input file to host grid and copy over device grid
  read_input(options.input_file, h_grid);
  hipMemcpy(d_grid, h_grid, grid_bytes, hipMemcpyHostToDevice);

  // Define block size as well as number of blocks
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size(options.size / BLOCK_SIZE, options.size / BLOCK_SIZE);

  for (int i = 0; i < options.loops; ++i) {
    evolution<<<grid_size, block_size>>>(d_grid, d_next_gen_grid, options.size);
    // Swap grids
    d_tmp_grid = d_grid;
    d_grid = d_next_gen_grid;
    d_next_gen_grid = d_tmp_grid;
  }

  // Copy results back to host grid
  hipMemcpy(h_grid, d_grid, grid_bytes, hipMemcpyDeviceToHost);

  // Free resources
  hipFree(d_grid);
  hipFree(d_next_gen_grid);
  free(h_grid);

  return 0;
}

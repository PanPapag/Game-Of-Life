#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define BLOCK_SIZE 4

typedef struct options {
  int loops;
  int size;
  char* input_file;
  char* output_file;
} program_options;

program_options options;
options.input_file = NULL;
options.output_file = NULL;

void parse_command_line_arguments(int argc, char* argv[]) {
  for (int i = 1; i < argc; i++) {
    if (!strcmp(argv[i], "-l")) {
      options.loops = atoi(argv[i+1]);
    } else if (!strcmp(argv[i], "-n")) {
      options.size = atoi(argv[i+1]);
    } else if (!strcmp(argv[i], "-i")) {
      options.input_file = strdup(argv[i+1]);
    } else if (!strcmp(argv[i], "-o")) {
      options.output_file = strdup(argv[i+1]);
    }
  }
}

__device__ char get_cell_inbounds(int x, int y, int size, char* grid) {
  if (x >= 0 && x < size && y >= 0 && y < size) {
    return grid[x * size + y];
  }
  return '0';
}

__device__ int count_alive_neighbours(int x, int y, int size, char* grid) {
  int alive_neighbours = (get_cell_inbounds(x-1, y, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y, size, grid) == '1') +
                         (get_cell_inbounds(x, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x, y+1, size, grid) == '1') +
                         (get_cell_inbounds(x-1, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x-1, y+1, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y-1, size, grid) == '1') +
                         (get_cell_inbounds(x+1, y+1, size, grid) == '1');
  return alive_neighbours;
}

__device__ void apply_game_rules(int index, char* cur_grid, char* next_grid, int alive_neighbours) {
  if (cur_grid[index] == '1') {
    // 0 or 1 neighbours -> the cell dies
    if (alive_neighbours < 2) {
      next_grid[index] = '0';
    }
    // 2 or 3 neighbours -> the cell survives
    else if (alive_neighbours < 4) {
      next_grid[index] = '1';
    }
    // more than 4 neighbours -> the cell dies due to overpopulation
    else {
      next_grid[index] = '0';
    }
  }
  // rules regarding dead cells
  else {
    // exactly 3 neighbours -> a new cell is born
    if (alive_neighbours == 3) {
      next_grid[index] = '1';
    } else {
      next_grid[index] = '0';
    }
  }
}

__global__ void evolution(char* cur_grid, char* next_grid, int size) {
  int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  int index = x * size + y;

  int alive_neighbours = count_alive_neighbours(x, y, size, cur_grid);

  apply_game_rules(index, cur_grid, next_grid, alive_neighbours);
}

void read_input(const char* input_file, char* grid) {
  FILE *fp;
  int i = 0;
  char c;
  fp = fopen(input_file, "r");
  while ((c = (char) fgetc(fp)) != '\n') {
    grid[i++] = c;
  }
}

int main(int argc, char* argv[]) {
  char* h_grid;            // Grid on host
  char* d_grid;            // Grid on device
  char* d_next_gen_grid;   // Next generation grid used on device
  char* d_tmp_grid;        // tmp grid pointer used to switch between grid and next_gen_grid
  float time;
  hipEvent_t start, stop;

  parse_command_line_arguments(argc, argv);

  size_t grid_bytes = options.size * options.size * sizeof(char);

  // Allocate memory for host grid
  h_grid = (char*) malloc(grid_bytes);

  // Allocate memory for device grids
  hipMalloc((void **)&d_grid, grid_bytes);
  hipMalloc((void **)&d_next_gen_grid, grid_bytes);

  // Read input file to host grid and copy over device grid
  read_input(options.input_file, h_grid);
  hipMemcpy(d_grid, h_grid, grid_bytes, hipMemcpyHostToDevice);

  // Define block size as well as number of blocks
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_size(options.size / BLOCK_SIZE, options.size / BLOCK_SIZE);

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i < options.loops; ++i) {
    evolution<<<grid_size, block_size>>>(d_grid, d_next_gen_grid, options.size);
    d_tmp_grid = d_grid;
    d_grid = d_next_gen_grid;
    d_next_gen_grid = d_tmp_grid;
    hipMemcpy(h_grid, d_next_gen_grid, grid_bytes, hipMemcpyDeviceToHost);
  }

  // Copy results back to host grid
  hipMemcpy(h_grid, d_grid, grid_bytes, hipMemcpyDeviceToHost);

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

  printf("Time elapsed:  %3.1f ms\n", time);

  // Free resources
  hipFree(d_grid);
  hipFree(d_next_gen_grid);
  free(h_grid);
  if (options.input_file != NULL) {
    free(options.input_file);
  }
  if (options.output_file != NULL) {
    free(options.output_file);
  }

  return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../../common/headers/io_utils.h"
#include "../../common/headers/utils.h"

#define BLOCK_SIZE 4

extern program_options options;

__device__ char get_cell_inbounds(int x, int y, char* grid) {
  if (x >=0 && x < options.size && y >= 0 && y < options.size) {
    return grid[x * options.size + y];
  }
  return '0';
}

__device__ void count_alive_neighbors(int x, int y, char* cur_grid) {
  int alive_neighbors = 0;

  alive_neighbors += 

}

__global__ void evolution(char* cur_grid, char* next_grid) {
  int x = blockDim.x * BLOCK_SIZE + threadIdx.x;
  int y = blockDim.y * BLOCK_SIZE + threadIdx.y;
  int index = x * options.size + y;

  int alive_neighbors = count_alive_neighbors(x, y, cur_grid);
}

void read_input(const char* input_file, char* grid) {
  int i = 0;
  char c;
  while ((c = getchar()) != '\n') {
    grid[i++] = c;
  }
}

int main(int argc, char* argv[]) {
  char* h_grid;            // Grid on host
  char* d_grid;            // Grid on device
  char* d_next_gen_grid;   // Next generation grid used on device
  char* d_tmp_grid;        // tmp grid pointer used to switch between grid and next_gen_grid

  parse_command_line_arguments(argc, argv);

  size_t grid_bytes = options.size * options.size * sizeof(char);

  // Allocate memory for host grid
  h_grid = (char*) malloc(grid_bytes);

  // Allocate memory for device grids
  hipMalloc(&d_grid, grid_bytes);
  hipMalloc(&d_next_gen_grid, grid_bytes);

  // Read input file to host grid and copy over device grid
  read_input(options.input_file, h_grid);
  hipMemcpy(d_grid, h_grid, grid_bytes, hipMemcpyHostToDevice);

  // Define block size as well as number of blocks
  dim3 block_size(BLOCK_SIDE, BLOCK_SIDE);
	dim3 grid_size(options.size / BLOCK_SIZE, options.size / BLOCK_SIZE);

  for (int i = 0; i < options.loops; ++i) {

    // Swap grids
    d_tmp_grid = d_grid;
    d_grid = d_next_gen_grid;
    d_next_gen_grid = d_tmp_rid;
  }

  // Copy results back to host grid
  hipMemcpy(h_grid, d_grid, grid_bytes, hipMemcpyDeviceToHost);

  // Free resources
  hipFree(d_grid);
  hipFree(d_next_gen_grid);
  free(h_grid);

  return 0;
}
